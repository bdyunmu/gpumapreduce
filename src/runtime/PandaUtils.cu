/*

        Copyright 2012 The Trustees of Indiana University. All rights reserved.
        Panda: co-processing SPMD computations on GPUs and CPUs.

        File: PandaUtils.cu
        First Version:          2012-07-01 V0.1
        Last UPdates:           2018-04-28 v0.61
        Developer: Hui Li (huili@ruijie.com.cn)
*/

#include "Panda.h"
#include <unistd.h> 
#include <sys/time.h>
#include <memory>
#include <stdio.h>
#ifndef __PANDA_UTILS_CU__
#define __PANDA_UTILS_CU__

namespace panda{

void getGPUDevProp(){
	int devCount = 0;
	hipError_t error_id = hipGetDeviceCount(&devCount);
	if(error_id != hipSuccess){
	printf("Result = FAIL\n");
	exit(-1);
	}
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t gpu_dev;
	hipGetDeviceProperties(&gpu_dev,dev);
	printf("GPU Total amount of global memory: %.0f Mbytes\n",(float)gpu_dev.totalGlobalMem/1048576.0f);
	printf("GPU Max Clock rate: %.0f GHz\n",gpu_dev.clockRate*1e-6f);
	printf("  Memory Clock rate:  %.0f Mhz\n",gpu_dev.memoryClockRate*1e-3f);
	printf("  Memory Bus Width:  %d-bit\n",gpu_dev.memoryBusWidth);
}

double getGPUMemBandwidthGb(){
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t gpu_dev;
	hipGetDeviceProperties(&gpu_dev,dev);
	double gghz = gpu_dev.memoryClockRate*1e-6f;
	int gbit = gpu_dev.memoryBusWidth;
	double gmbd = gghz*gbit/8*4;
	return gmbd; //in GB/s
}
double getGPUMemSizeGb(){
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t gpu_dev;
	hipGetDeviceProperties(&gpu_dev,dev);
	return (gpu_dev.totalGlobalMem/1048576.0f/1024.0);
}
double getGPUGHz(){
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t gpu_dev;
	hipGetDeviceProperties(&gpu_dev,dev);
	return (gpu_dev.clockRate*1e-6f);
}
int getGPUCoresNum() { 
	int arch_cores_sm[3] = {1, 8, 32 };
	hipDeviceProp_t gpu_dev;
	int tid = 0;
	hipGetDeviceProperties(&gpu_dev,tid);
	int sm_per_multiproc = 1;
	if (gpu_dev.major == 9999 && gpu_dev.minor == 9999)
			sm_per_multiproc = 1;
	else if (gpu_dev.major <=2)
			sm_per_multiproc = arch_cores_sm[gpu_dev.major];
	else
			sm_per_multiproc = arch_cores_sm[2];
	ShowLog("pgc Configure Device ID:%d: Device Name:%s MultProcessorCount:%d sm_per_multiproc:%d", 0, gpu_dev.name,gpu_dev.multiProcessorCount,sm_per_multiproc);
	return ((gpu_dev.multiProcessorCount)*(sm_per_multiproc));
}

void sleep(int sleepMs)
{
#ifdef __linux
    usleep(sleepMs * 1000);   // usleep takes sleep time in us
#endif
#ifdef _WIN32
    Sleep(sleepMs);
#endif
}



int getCPUCoresNum() { 

#ifdef WIN32 
    SYSTEM_INFO sysinfo; 
    GetSystemInfo(&sysinfo); 
    return sysinfo.dwNumberOfProcessors; 
#elif MACOS 
    int nm[2]; 
    size_t len = 4; 
    uint32_t count; 
 
    nm[0] = CTL_HW; nm[1] = HW_AVAILCPU; 
    sysctl(nm, 2, &count, &len, NULL, 0); 
 
    if(count < 1) { 
        nm[1] = HW_NCPU; 
        sysctl(nm, 2, &count, &len, NULL, 0); 
        if(count < 1) { count = 1; } 
    } 
    return count; 
#elif __linux
    return sysconf(_SC_NPROCESSORS_ONLN); 
#endif 

}
double getCPUMemSizeGb(){
	char cmd[128];
	sprintf(cmd,"cat /proc/meminfo |grep MemTotal|awk -F:' ' '{print $2}'");
	FILE *fp = popen(cmd,"r");
	if(fp == NULL){
		printf("cat /proc/meminfo/ fp ==  NULL\n");
		exit(0);
	}//if
	char buf1[128];
	fread(buf1,128,1,fp);
	pclose(fp);
	int cpuMemSize;
	char buf2[128];
	sscanf(buf1,"%d %s",&cpuMemSize,buf2);
	return cpuMemSize/1024/1024;
}

double getCPUMemBandwidthGb(){
	char cmd[128];
	sprintf(cmd,"dmidecode -t memory|grep \"Type\"|grep -v \"Type Detail\" |grep -v \"Correction Type\"|uniq|awk -F':' '{print $2}'");
	FILE *fp = popen(cmd,"r");
	if(fp == NULL){
		printf("dmidecode -t memory == NULL\n");
		exit(0);
	}
	char output[128];
	fread(output,128,1,fp);
	pclose(fp);
	int multiplier = 1;
	if(strstr(output,"DDR3")){
		multiplier = 8;
	}else if(strstr(output,"DDR2")){
		multiplier = 4;
	}else if(strstr(output,"DDR")){
		multiplier = 2;
	}
	sprintf(cmd,"dmidecode -t memory|grep \"Data Width\"|uniq|awk -F':' '{print $2}'");	
	fp = popen(cmd,"r");
	if(fp == NULL){
		printf("dmidecode -t memory == NULL\n");
		exit(0);
	}
	fread(output,128,1,fp);
	pclose(fp);
	int memBits = 32;
	char buf2[128];
	sscanf(output,"%d %s",&memBits,buf2);
	sprintf(cmd,"dmidecode -t memory|grep Speed|grep -v \"Unknow\"|grep -v \"Configured Clock Speed\"|uniq|awk -F':' '{print $2}'");	
	fp = popen(cmd,"r");
	if(fp == NULL){
		printf("dmidecode -t memory == NULL\n");
		exit(0);
	}
	fread(output,128,1,fp);
	pclose(fp);
	int memSpeed = 1000;
	sscanf(output,"%d %s",&memSpeed,buf2);
	double memBandwidth = 0.0;
	memBandwidth = memSpeed/8.0*memBits*multiplier/8.0/1024.0;
	return memBandwidth;
}

double getCPUGHz(){
	char cmd[128];
	sprintf(cmd,"cat /proc/cpuinfo |grep MHz|awk -F':' '{print $2}'|head -n 1");
	FILE *fp = popen(cmd,"r");
	if(fp == NULL){
		printf("cat /proc/cpuinfo/ fp == NULL\n");
		exit(0);
	}
	char buf[128];
	fread(buf,128,1,fp);
	pclose(fp);
	double CPUGHz = 0;
	sscanf(buf,"%lf",&CPUGHz);
	return CPUGHz/1000.0;
}
double PandaTimer(){
	#ifndef _WIN32
	static struct timeval tv;
	gettimeofday(&tv,NULL);
	double curTime = tv.tv_sec + tv.tv_usec/1000000.0;
	return curTime;
	#else
	double curTime = GetTickCount(); 
	curTime /=1000.0;
	return curTime;
	#endif
}//double PandaTimer()

void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "[PandaError][%s][%i]: CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
		exit((int)err);        
	}
}

}
#endif //__PANDA_UTILS_CU__
